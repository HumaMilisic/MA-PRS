#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust\device_vector.h>
#include <thrust\copy.h>
#include <thrust\reduce.h>
#include <thrust\fill.h>
#include <thrust\device_ptr.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <string>
#include <iterator>
#include <queue>
#include <ctime>
#include <algorithm>
#define F(x) cout<<#x " = "<<x<<endl;

using namespace std;

long strToBr(string line)
{
	long br = 0;
	long size = line.size();
	//F(size);
	//F(pow(10, 3));
	for (long i = 0; i < line.size(); i++)
	{
		char a = line[i];
		a -= '0';
		//F((long)a);
		br += a*pow(10, size - 1);
		size--;
	}
	//F(br);
	br--;
	return br;
}

vector<string> token(string line)
{
	vector<string>tempNiz;
	string temp = "";
	for (long i = 0; i < line.size(); i++)
	{
		if (line[i] == ' ')
		{
			tempNiz.push_back(temp);
			temp = "";
			continue;
		}
		temp += line[i];
	}
	//if (temp != "")
	tempNiz.push_back(temp);
	return tempNiz;
}

vector<long> strURedBrojeva(string line)
{
	vector<long>rez;
	vector<string>temp = token(line);
	for (long i = 0; i < temp.size(); i++)
	{
		//if (temp[i] != "")
		int br = strToBr(temp[i]);
		if (br >= 0)	rez.push_back(br);
	}
	return rez;
}

__global__
void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		C[i] = A[i] + B[i];
	}
}

__global__
void prazno(const long *V,long sizeV,const long*E,long sizeE,long *F,long*X,long*C)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < sizeV)
	{
		if (F[i])
		{
			F[i] = false;
			X[i] = true;
			long pozP = V[i],
				pozK = i + 1 < sizeV ? V[i + 1] : sizeE;
			for (long j = pozP; j < pozK; j++)
			{
				long susjed = E[j];
				if (!X[susjed])
				{
					F[susjed] = 1;
					C[susjed] = C[i] + 1;
				}
			}
		}
	}
	//printf("%d\n", i);
}

void sekvencijalno(const vector<bool>&graf, int n, long i=0)
{
	queue<long>redCvorova;
	vector<long>redoslijed;
	vector<bool>posjecenCvor;
	vector<bool>dodanURedCvor;
	posjecenCvor.resize(n, false);
	dodanURedCvor.resize(n, false);
	redCvorova.push(0);
	dodanURedCvor[0] = true;

	clock_t t1;
	t1 = clock();
	long brPosjecenih = 0;
	redoslijed.push_back(0);
	while (redCvorova.size() > 0 && brPosjecenih < n)
	{
		long trenutniCvor = redCvorova.front();
		redCvorova.pop();
		brPosjecenih++;

		if (posjecenCvor[trenutniCvor])
			continue;

		posjecenCvor[trenutniCvor] = true;
		//i = trenutni cvor
		for (int j = 0; j < n; j++)
		{
			int index = trenutniCvor*n + j;
			if (!graf[index])
				continue;
			if (posjecenCvor[j] || dodanURedCvor[j])
				continue;
			redCvorova.push(j);
			dodanURedCvor[j] = true;
			redoslijed.push_back(j);
			//if (graf[index])
		}
	}
	double diff = (double)(clock() - t1) / CLOCKS_PER_SEC;
	cout << endl << i << ": The time taken for Breadth first search: " << diff << endl;
	//FFC(redoslijed)
}


int main()
{
	/*size_t N = 10;
	int *raw_ptr;
	hipMalloc((void**)&raw_ptr,N*sizeof(int));
	thrust::device_ptr<int> dev_ptr(raw_ptr);
	thrust::fill(dev_ptr, dev_ptr + N, (int)1);
	int rez = thrust::reduce(dev_ptr, dev_ptr + N);
	F(rez);
	char t = getchar();
	*/
	
	//input grafa

	long *h_V(NULL), *h_E(NULL),// *d_V(NULL), *d_E(NULL),
		n(0), m(0);

	//int *h_F(NULL), *h_X(NULL), *h_C(NULL);

	size_t sizeV(0), sizeE(0);

	cout << "Input grafa\n";
	ifstream in("_input/simple.prs");
	//ifstream in("_input/luxembourg.osm.graph");
	//ifstream in("_input/ecology1.graph");
	//ifstream in("input/kron_g500-simple-logn21.graph");
	string line;
	vector<long>tempV, tempE;
	bool flagUnm = false;

	long brI = 0, brJ = 0;
	std::cout << "Input started: " << endl;
	while (getline(in, line))
	{
		if (line[0] == '%')
			continue;

		if (!flagUnm)
		{
			flagUnm = true;
			vector<long> temp(strURedBrojeva(line));
			n = temp[0] + 1;
			m = temp[1] + 1;

			//sizeV = n * sizeof(long);
			//sizeE = m * sizeof(long);
			tempV.reserve(n);
			tempE.reserve(m);
			tempV.push_back(0);
			brI++;
			//h_V[brI++] = 0;
			continue;
		}
		vector<long> temp = strURedBrojeva(line);
		for (long i = 0; i < temp.size(); i++)
		{
			tempE.push_back(temp[i]);
			brJ++;
		}
		if (tempV.size()<n)
			tempV.push_back(tempE.size());
	}

	sizeV = tempV.size()*sizeof(long); n = tempV.size();
	sizeE = tempE.size()*sizeof(long); m = tempE.size();

	h_V = (long *)malloc(sizeV);
	std::copy(tempV.begin(), tempV.end(), h_V);
	tempV.clear();

	h_E = (long *)malloc(sizeE);
	std::copy(tempE.begin(), tempE.end(), h_E);
	tempE.clear();

	in.close();
	cout << "Input gotov" << endl;

	//alokacija h_F, h_X i h_C
	cout << "Alokacija host\n";
	long *h_F(NULL), *h_X(NULL), *h_C(NULL);
	h_F = (long*)malloc(sizeV);
	h_X = (long*)malloc(sizeV);
	h_C = (long*)malloc(sizeV);
	memset(h_F, 0, sizeV);
	memset(h_X, 0, sizeV);
	memset(h_C, 127, sizeV);

	//pocetne postavke za BFS
	long pocetniCvor = 0;
	h_F[pocetniCvor] = 1;
	h_C[pocetniCvor] = 0;

	//alokacija na device
	cout << "Alokacija device\n";
	long *d_E(NULL), *d_V(NULL), 
		*d_F(NULL), *d_X(NULL), *d_C(NULL);
	hipMalloc((void**)&d_E, sizeE);
	hipMalloc((void**)&d_V, sizeV);
	hipMalloc((void**)&d_F, sizeV);
	hipMalloc((void**)&d_X, sizeV);
	hipMalloc((void**)&d_C, sizeV);


	//kopiranje na device
	cout << "Kopiranje na device\n";
	hipMemcpy(d_E, h_E, sizeE, hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeV, hipMemcpyHostToDevice);
	hipMemcpy(d_F, h_F, sizeV, hipMemcpyHostToDevice);
	hipMemcpy(d_X, h_X, sizeV, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeV, hipMemcpyHostToDevice);

	//potrebno za BFS
	thrust::device_ptr<long> dev_ptr(d_F);
	hipDeviceSynchronize();
	cout << "pocinje BFS paralelni\n";
	while (thrust::reduce(dev_ptr, dev_ptr + n))
	{
		//cout << "While petlja\n";
		int threadsPerBlock = 256<n?256:n;
		int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

		prazno <<<blocksPerGrid,threadsPerBlock >>>(d_V,sizeV,d_E,sizeE,d_F,d_X,d_C);
		//vectorAdd << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, numElements);

		//break;
		hipDeviceSynchronize();
	}
	cout << "Copy C to host\n";
	hipMemcpy(h_C, d_C, sizeV, hipMemcpyDeviceToHost);
	//copy(h_C, h_C + n, ostream_iterator<long>(cout, ","));

	cout << "\nOslobadjanje host\n";
	free(h_E);
	free(h_V);
	free(h_F);
	free(h_X);
	free(h_C);

	cout << "Oslobadjanje device\n";
	hipFree(d_E);
	hipFree(d_V);
	hipFree(d_F);
	hipFree(d_X);
	hipFree(d_C);
	cout << "\nDone!!!!!!!!!!!!!" << endl;
	char t = getchar();
    return 0;
}

