#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust\device_vector.h>
#include <thrust\copy.h>
#include <thrust\reduce.h>
#include <thrust\fill.h>
#include <thrust\device_ptr.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <string>
#include <iterator>
#include <queue>
#include <ctime>
#include <algorithm>
#include <climits>
#include "hip/hip_runtime_api.h"
#define F(x) cout<<#x " = "<<x<<endl;

using namespace std;

__device__  __managed__ bool done = true;
__device__ __managed__  long iteration = 0;

ofstream _log1("costArrayProvjeraDebug.log");

__global__
void postaviGlobalneLol(bool &done, long &iteration,bool vdone,long viter)
{
	done = vdone;
	iteration = viter;
}

__global__
void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		C[i] = A[i] + B[i];
	}
}

__global__
void prazno(const long *V, long sizeV, const long*E, long sizeE, long *F, long*X, long*C) // V - vertice, E - susjedi, F - za posjetiti, X posjeceno, C cost do ture
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < sizeV)
	{
		if (F[i])
		{
			F[i] = false;
			X[i] = true;
			long pozP = V[i],
				pozK = i + 1 < sizeV ? V[i + 1] : sizeE;
			for (long j = pozP; j < pozK; j++)
			{
				long susjed = E[j];
				if (!X[susjed])
				{
					F[susjed] = 1;
					C[susjed] = C[i] + 1;
				}
			}
		}
	}
	//printf("%d\n", i);
}

__global__
void kernel_1(const long *V, long sizeV, const long*E, long sizeE, long*C)/*long *F, long*X, */
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < sizeV)
	{
		if (C[i] == iteration)
		{
			//if (done)
			done = false;
			long pozP = V[i],
				pozK = i + 1 < sizeV ? V[i + 1] : sizeE;
			for (long j = pozP; j < pozK; j++)
			{
				long susjed = E[j];
				if (C[susjed]>iteration)
					C[susjed] = iteration + 1;
				//long j = C[susjed];
			}
		}
	}
}
//void kernel_1_Share(const long *V, long sizeV, const long*E, long sizeE, long*C)/*long *F, long*X, */
//{
//	//bool lokalDone = done;
//	long lokalIte = iteration;
//	int i = blockDim.x * blockIdx.x + threadIdx.x;
//	if (i < sizeV)
//	{
//		if (C[i] == lokalIte)
//		{
//			//if (done)
//			//if (lokalDone)
//				done = false;
//			//int a = atomicAnd(&done, false);
//			long pozP = V[i],
//				pozK = i + 1 < sizeV ? V[i + 1] : sizeE;
//			for (long j = pozP; j < pozK; j++)
//			{
//				long susjed = E[j];
//				if (C[susjed]>lokalIte)
//					C[susjed] = lokalIte + 1;
//				//long j = C[susjed];
//			}
//		}
//	}
//
//}

__global__
void BFSMainKernel(const long *V, long sizeV, const long *E, long sizeE)
{

}

extern "C" void uhvatiNiz(long *niz, long size)
{
	cout << "uhvati niz\n";
	copy(niz, niz + size, ostream_iterator<long>(cout, ", "));
}

extern "C" double paralelniBFS(long *h_V, long *h_E, long sizeV, long sizeE)
{
	cout << "paralelniBFS" << endl;
	
	cout << "Alokacija host\n";
	long *h_F(NULL), *h_X(NULL), *h_C(NULL);
	h_F = (long*)malloc(sizeV*sizeof(long));
	h_X = (long*)malloc(sizeV*sizeof(long));
	h_C = (long*)malloc(sizeV*sizeof(long));
	memset(h_F, 0, sizeV*sizeof(long));
	memset(h_X, 0, sizeV*sizeof(long));
	memset(h_C, 127, sizeV*sizeof(long));

	//pocetne postavke za BFS
	long pocetniCvor = 0;
	h_F[pocetniCvor] = 1;
	h_C[pocetniCvor] = 0;
	
	//alokacija na device
	cout << "Alokacija device\n";
	long *d_E(NULL), *d_V(NULL),
		*d_F(NULL), *d_X(NULL), *d_C(NULL);
	hipMalloc((void**)&d_E, sizeE*sizeof(long));
	hipMalloc((void**)&d_V, sizeV*sizeof(long));
	hipMalloc((void**)&d_F, sizeV*sizeof(long));
	hipMalloc((void**)&d_X, sizeV*sizeof(long));
	hipMalloc((void**)&d_C, sizeV*sizeof(long));

	//kopiranje na device
	cout << "Kopiranje na device\n";
	hipMemcpy(d_E, h_E, sizeE*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_F, h_F, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_X, h_X, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeV*sizeof(long), hipMemcpyHostToDevice);


	//potrebno za BFS
	thrust::device_ptr<long> dev_ptr(d_F);
	//bool paralelno = true, seq = false;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipDeviceSynchronize();

	cout << "pocinje BFS paralelni\n";
	clock_t p1;
	p1 = clock();
	hipEventRecord(start);
	while (thrust::reduce(dev_ptr, dev_ptr + sizeV))
	{
		//cout << "While petlja\n";
		int threadsPerBlock = 256<sizeV ? 256 : sizeV;
		int blocksPerGrid = (sizeV + threadsPerBlock - 1) / threadsPerBlock;
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
		
		prazno << <blocksPerGrid, threadsPerBlock >> >(d_V, sizeV, d_E, sizeE, d_F, d_X, d_C);
		
		//vectorAdd << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, numElements);
		//break;
		//hipDeviceSynchronize();
	}
	hipEventRecord(stop);
	double diff = (double)(clock() - p1) / CLOCKS_PER_SEC;
	//hipDeviceSynchronize();
	hipEventSynchronize(stop);
	float milliseconds = 0;
	double sec;
	hipEventElapsedTime(&milliseconds, start, stop);
	sec = milliseconds / 1000.0;
	//_log1  << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": event: " << sec << " s\n";
	cout << "Copy C to host\n";
	hipMemcpy(h_C, d_C, sizeV*sizeof(long), hipMemcpyDeviceToHost);
	_log1 << " ; " << endl;
	copy(h_C, h_C + sizeV, ostream_iterator<long>(_log1, " "));
	_log1 << endl;

	cout << "Oslobadjanje memorije\n";
	hipFree(d_E);
	hipFree(d_V);
	hipFree(d_F);
	hipFree(d_X);
	hipFree(d_C);

	free(h_F);
	free(h_X);
	free(h_C);

	return sec;
}

extern "C" double paralelniBFS_64(long *h_V, long *h_E, long sizeV, long sizeE)
{
	cout << "paralelniBFS" << endl;

	cout << "Alokacija host\n";
	long *h_F(NULL), *h_X(NULL), *h_C(NULL);
	h_F = (long*)malloc(sizeV*sizeof(long));
	h_X = (long*)malloc(sizeV*sizeof(long));
	h_C = (long*)malloc(sizeV*sizeof(long));
	memset(h_F, 0, sizeV*sizeof(long));
	memset(h_X, 0, sizeV*sizeof(long));
	memset(h_C, 127, sizeV*sizeof(long));

	//pocetne postavke za BFS
	long pocetniCvor = 0;
	h_F[pocetniCvor] = 1;
	h_C[pocetniCvor] = 0;

	//alokacija na device
	cout << "Alokacija device\n";
	long *d_E(NULL), *d_V(NULL),
		*d_F(NULL), *d_X(NULL), *d_C(NULL);
	hipMalloc((void**)&d_E, sizeE*sizeof(long));
	hipMalloc((void**)&d_V, sizeV*sizeof(long));
	hipMalloc((void**)&d_F, sizeV*sizeof(long));
	hipMalloc((void**)&d_X, sizeV*sizeof(long));
	hipMalloc((void**)&d_C, sizeV*sizeof(long));

	//kopiranje na device
	cout << "Kopiranje na device\n";
	hipMemcpy(d_E, h_E, sizeE*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_F, h_F, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_X, h_X, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeV*sizeof(long), hipMemcpyHostToDevice);


	//potrebno za BFS
	thrust::device_ptr<long> dev_ptr(d_F);
	//bool paralelno = true, seq = false;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipDeviceSynchronize();

	cout << "pocinje BFS paralelni\n";
	clock_t p1;
	p1 = clock();
	hipEventRecord(start);
	while (thrust::reduce(dev_ptr, dev_ptr + sizeV))
	{
		//cout << "While petlja\n";
		int m = 65;
		int threadsPerBlock = m<sizeV ? m : sizeV;
		int blocksPerGrid = (sizeV + threadsPerBlock - 1) / threadsPerBlock;
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

		prazno << <blocksPerGrid, threadsPerBlock >> >(d_V, sizeV, d_E, sizeE, d_F, d_X, d_C);

		//vectorAdd << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, numElements);
		//break;
		//hipDeviceSynchronize();
	}
	hipEventRecord(stop);
	double diff = (double)(clock() - p1) / CLOCKS_PER_SEC;
	//hipDeviceSynchronize();
	hipEventSynchronize(stop);
	float milliseconds = 0;
	double sec;
	hipEventElapsedTime(&milliseconds, start, stop);
	sec = milliseconds / 1000.0;
	//_log1  << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": event: " << sec << " s\n";
	//cout << "Copy C to host\n";
	//hipMemcpy(h_C, d_C, sizeV*sizeof(long), hipMemcpyDeviceToHost);
	//copy(h_C, h_C + sizeV, ostream_iterator<long>(cout, ","));


	cout << "Oslobadjanje memorije\n";
	hipFree(d_E);
	hipFree(d_V);
	hipFree(d_F);
	hipFree(d_X);
	hipFree(d_C);

	free(h_F);
	free(h_X);
	free(h_C);

	return sec;
}



extern "C" double paralelniBFS_1(long *h_V, long *h_E, long sizeV, long sizeE)
{
	cout << "paralelniBFS" << endl;

	cout << "Alokacija host\n";
	long *h_F(NULL), *h_X(NULL), *h_C(NULL);
	h_C = (long*)malloc(sizeV*sizeof(long));
	memset(h_C, 127, sizeV*sizeof(long));

	//pocetne postavke za BFS
	long pocetniCvor = 0;
	//h_F[pocetniCvor] = 1;
	h_C[pocetniCvor] = 0;

	//alokacija na device
	cout << "Alokacija device\n";
	long *d_E(NULL), *d_V(NULL),
		*d_F(NULL), *d_X(NULL), *d_C(NULL);
	hipMalloc((void**)&d_E, sizeE*sizeof(long));
	hipMalloc((void**)&d_V, sizeV*sizeof(long));
	hipMalloc((void**)&d_C, sizeV*sizeof(long));

	//kopiranje na device
	cout << "Kopiranje na device\n";
	hipMemcpy(d_E, h_E, sizeE*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeV*sizeof(long), hipMemcpyHostToDevice);


	//potrebno za BFS
	iteration = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipDeviceSynchronize();

	cout << "pocinje BFS paralelni\n";
	clock_t p1;
	p1 = clock();
	hipEventRecord(start);
	do
	{
		done = true;
		int threadsPerBlock = 256<sizeV ? 256 : sizeV;
		int blocksPerGrid = (sizeV + threadsPerBlock - 1) / threadsPerBlock;
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

		kernel_1 << <blocksPerGrid, threadsPerBlock >> >(d_V, sizeV, d_E, sizeE, d_C);
		
		hipDeviceSynchronize();
		iteration++;
	} while (!done);
	hipEventRecord(stop);
	double diff = (double)(clock() - p1) / CLOCKS_PER_SEC;
	//hipDeviceSynchronize();
	hipEventSynchronize(stop);
	float milliseconds = 0;
	double sec;
	hipEventElapsedTime(&milliseconds, start, stop);
	sec = milliseconds / 1000.0;
	//_log1  << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": event: " << sec << " s\n";
	cout << "Copy C to host\n";
	hipMemcpy(h_C, d_C, sizeV*sizeof(long), hipMemcpyDeviceToHost);
	_log1 << " ; "<< endl;
	copy(h_C, h_C + sizeV, ostream_iterator<long>(_log1, " "));
	_log1 <<  endl;

	cout << "Oslobadjanje memorije\n";
	hipFree(d_E);
	hipFree(d_V);
	hipFree(d_C);

	free(h_C);
	cout << "done" << endl;
	_log1.close();
	return sec;
}

//extern "C" double paralelniBFS_1_Share(long *h_V, long *h_E, long sizeV, long sizeE)
//{
//	cout << "paralelniBFS" << endl;
//
//	cout << "Alokacija host\n";
//	long *h_F(NULL), *h_X(NULL), *h_C(NULL);
//	h_C = (long*)malloc(sizeV*sizeof(long));
//	memset(h_C, 127, sizeV*sizeof(long));
//
//	//pocetne postavke za BFS
//	long pocetniCvor = 0;
//	//h_F[pocetniCvor] = 1;
//	h_C[pocetniCvor] = 0;
//
//	//alokacija na device
//	cout << "Alokacija device\n";
//	long *d_E(NULL), *d_V(NULL),
//		*d_F(NULL), *d_X(NULL), *d_C(NULL);
//	hipMalloc((void**)&d_E, sizeE*sizeof(long));
//	hipMalloc((void**)&d_V, sizeV*sizeof(long));
//	hipMalloc((void**)&d_C, sizeV*sizeof(long));
//
//	//kopiranje na device
//	cout << "Kopiranje na device\n";
//	hipMemcpy(d_E, h_E, sizeE*sizeof(long), hipMemcpyHostToDevice);
//	hipMemcpy(d_V, h_V, sizeV*sizeof(long), hipMemcpyHostToDevice);
//	hipMemcpy(d_C, h_C, sizeV*sizeof(long), hipMemcpyHostToDevice);
//
//
//	//potrebno za BFS
//	iteration = 0;
//	hipEvent_t start, stop;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//	hipDeviceSynchronize();
//
//	cout << "pocinje BFS paralelni\n";
//	clock_t p1;
//	p1 = clock();
//	hipEventRecord(start);
//	do
//	{
//		done = true;
//		int threadsPerBlock = 256<sizeV ? 256 : sizeV;
//		int blocksPerGrid = (sizeV + threadsPerBlock - 1) / threadsPerBlock;
//		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
//		//kernel_1_Share<< <blocksPerGrid,threadsPerBlock> >>(d_V, sizeV, d_E, sizeE, d_C);
//		//kernel_1_Share << <blocksPerGrid, threadsPerBlock >> >(d_V, sizeV, d_E, sizeE, d_C);
//		
//		hipDeviceSynchronize();
//		iteration++;
//	} while (!done);
//	hipEventRecord(stop);
//	double diff = (double)(clock() - p1) / CLOCKS_PER_SEC;
//	//hipDeviceSynchronize();
//	hipEventSynchronize(stop);
//	float milliseconds = 0;
//	double sec;
//	hipEventElapsedTime(&milliseconds, start, stop);
//	sec = milliseconds / 1000.0;
//	//_log1  << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
//	cout << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
//	cout << endl << ": event: " << sec << " s\n";
//	cout << "Copy C to host\n";
//	hipMemcpy(h_C, d_C, sizeV*sizeof(long), hipMemcpyDeviceToHost);
//	_log1 << " ; "<< endl;
//	copy(h_C, h_C + sizeV, ostream_iterator<long>(_log1, " "));
//	_log1 <<  endl;
//
//	cout << "Oslobadjanje memorije\n";
//	hipFree(d_E);
//	hipFree(d_V);
//	hipFree(d_C);
//
//	free(h_C);
//	cout << "done" << endl;
//	_log1.close();
//	return sec;
//}