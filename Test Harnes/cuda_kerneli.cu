#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust\device_vector.h>
#include <thrust\copy.h>
#include <thrust\reduce.h>
#include <thrust\fill.h>
#include <thrust\device_ptr.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <string>
#include <iterator>
#include <queue>
#include <ctime>
#include <algorithm>
#include <climits>
#include "hip/hip_runtime_api.h"
#include <windows.h>
#include "kerneli.cuh"
#define F(x) cout<<#x " = "<<x<<endl;

using namespace std;


ofstream _log1("costArrayProvjeraDebug.log");



extern "C" void uhvatiNiz(long *niz, long size)
{
	cout << "uhvati niz\n";
	copy(niz, niz + size, ostream_iterator<long>(cout, ", "));
}

extern "C" double paralelniBFS(long *h_V, long *h_E, long sizeV, long sizeE)
{
	cout << "paralelniBFS" << endl;
	
	cout << "Alokacija host\n";
	long *h_F(NULL), *h_X(NULL), *h_C(NULL);
	h_F = (long*)malloc(sizeV*sizeof(long));
	h_X = (long*)malloc(sizeV*sizeof(long));
	h_C = (long*)malloc(sizeV*sizeof(long));
	memset(h_F, 0, sizeV*sizeof(long));
	memset(h_X, 0, sizeV*sizeof(long));
	memset(h_C, 127, sizeV*sizeof(long));

	//pocetne postavke za BFS
	long pocetniCvor = 0;
	h_F[pocetniCvor] = 1;
	h_C[pocetniCvor] = 0;
	
	//alokacija na device
	cout << "Alokacija device\n";
	long *d_E(NULL), *d_V(NULL),
		*d_F(NULL), *d_X(NULL), *d_C(NULL);
	hipMalloc((void**)&d_E, sizeE*sizeof(long));
	hipMalloc((void**)&d_V, sizeV*sizeof(long));
	hipMalloc((void**)&d_F, sizeV*sizeof(long));
	hipMalloc((void**)&d_X, sizeV*sizeof(long));
	hipMalloc((void**)&d_C, sizeV*sizeof(long));

	//kopiranje na device
	cout << "Kopiranje na device\n";
	hipMemcpy(d_E, h_E, sizeE*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_F, h_F, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_X, h_X, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeV*sizeof(long), hipMemcpyHostToDevice);


	//potrebno za BFS
	thrust::device_ptr<long> dev_ptr(d_F);
	//bool paralelno = true, seq = false;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipDeviceSynchronize();

	cout << "pocinje BFS paralelni\n";
	clock_t p1;
	p1 = clock();
	hipEventRecord(start);
	while (thrust::reduce(dev_ptr, dev_ptr + sizeV))
	{
		//cout << "While petlja\n";
		int threadsPerBlock = 256<sizeV ? 256 : sizeV;
		int blocksPerGrid = (sizeV + threadsPerBlock - 1) / threadsPerBlock;
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
		
		prazno << <blocksPerGrid, threadsPerBlock >> >(d_V, sizeV, d_E, sizeE, d_F, d_X, d_C);
		
		//vectorAdd << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, numElements);
		//break;
		//hipDeviceSynchronize();
	}
	hipEventRecord(stop);
	double diff = (double)(clock() - p1) / CLOCKS_PER_SEC;
	//hipDeviceSynchronize();
	hipEventSynchronize(stop);
	float milliseconds = 0;
	double sec;
	hipEventElapsedTime(&milliseconds, start, stop);
	sec = milliseconds / 1000.0;
	//_log1  << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": event: " << sec << " s\n";
	cout << "Copy C to host\n";
	hipMemcpy(h_C, d_C, sizeV*sizeof(long), hipMemcpyDeviceToHost);
	_log1 << " ; " << endl;
	copy(h_C, h_C + sizeV, ostream_iterator<long>(_log1, " "));
	_log1 << endl;

	cout << "Oslobadjanje memorije\n";
	hipFree(d_E);
	hipFree(d_V);
	hipFree(d_F);
	hipFree(d_X);
	hipFree(d_C);

	free(h_F);
	free(h_X);
	free(h_C);

	return sec;
}

extern "C" double paralelniBFS_64(long *h_V, long *h_E, long sizeV, long sizeE)
{
	cout << "paralelniBFS" << endl;

	cout << "Alokacija host\n";
	long *h_F(NULL), *h_X(NULL), *h_C(NULL);
	h_F = (long*)malloc(sizeV*sizeof(long));
	h_X = (long*)malloc(sizeV*sizeof(long));
	h_C = (long*)malloc(sizeV*sizeof(long));
	memset(h_F, 0, sizeV*sizeof(long));
	memset(h_X, 0, sizeV*sizeof(long));
	memset(h_C, 127, sizeV*sizeof(long));

	//pocetne postavke za BFS
	long pocetniCvor = 0;
	h_F[pocetniCvor] = 1;
	h_C[pocetniCvor] = 0;

	//alokacija na device
	cout << "Alokacija device\n";
	long *d_E(NULL), *d_V(NULL),
		*d_F(NULL), *d_X(NULL), *d_C(NULL);
	hipMalloc((void**)&d_E, sizeE*sizeof(long));
	hipMalloc((void**)&d_V, sizeV*sizeof(long));
	hipMalloc((void**)&d_F, sizeV*sizeof(long));
	hipMalloc((void**)&d_X, sizeV*sizeof(long));
	hipMalloc((void**)&d_C, sizeV*sizeof(long));

	//kopiranje na device
	cout << "Kopiranje na device\n";
	hipMemcpy(d_E, h_E, sizeE*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_F, h_F, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_X, h_X, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeV*sizeof(long), hipMemcpyHostToDevice);


	//potrebno za BFS
	thrust::device_ptr<long> dev_ptr(d_F);
	//bool paralelno = true, seq = false;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipDeviceSynchronize();

	cout << "pocinje BFS paralelni\n";
	clock_t p1;
	p1 = clock();
	hipEventRecord(start);
	while (thrust::reduce(dev_ptr, dev_ptr + sizeV))
	{
		//cout << "While petlja\n";
		int m = 65;
		int threadsPerBlock = m<sizeV ? m : sizeV;
		int blocksPerGrid = (sizeV + threadsPerBlock - 1) / threadsPerBlock;
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

		prazno << <blocksPerGrid, threadsPerBlock >> >(d_V, sizeV, d_E, sizeE, d_F, d_X, d_C);

		//vectorAdd << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, numElements);
		//break;
		//hipDeviceSynchronize();
	}
	hipEventRecord(stop);
	double diff = (double)(clock() - p1) / CLOCKS_PER_SEC;
	//hipDeviceSynchronize();
	hipEventSynchronize(stop);
	float milliseconds = 0;
	double sec;
	hipEventElapsedTime(&milliseconds, start, stop);
	sec = milliseconds / 1000.0;
	//_log1  << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": event: " << sec << " s\n";
	//cout << "Copy C to host\n";
	//hipMemcpy(h_C, d_C, sizeV*sizeof(long), hipMemcpyDeviceToHost);
	//copy(h_C, h_C + sizeV, ostream_iterator<long>(cout, ","));


	cout << "Oslobadjanje memorije\n";
	hipFree(d_E);
	hipFree(d_V);
	hipFree(d_F);
	hipFree(d_X);
	hipFree(d_C);

	free(h_F);
	free(h_X);
	free(h_C);

	return sec;
}



extern "C" double paralelniBFS_1(long *h_V, long *h_E, long sizeV, long sizeE)
{
	cout << "paralelniBFS" << endl;

	cout << "Alokacija host\n";
	long *h_C(NULL);
	h_C = (long*)malloc(sizeV*sizeof(long));
	memset(h_C, 127, sizeV*sizeof(long));

	//pocetne postavke za BFS
	long pocetniCvor = 0;
	//h_F[pocetniCvor] = 1;
	h_C[pocetniCvor] = 0;

	//alokacija na device
	cout << "Alokacija device\n";
	long *d_E(NULL), *d_V(NULL),*d_C(NULL);
	hipMalloc((void**)&d_E, sizeE*sizeof(long));
	hipMalloc((void**)&d_V, sizeV*sizeof(long));
	hipMalloc((void**)&d_C, sizeV*sizeof(long));

	//kopiranje na device
	cout << "Kopiranje na device\n";
	hipMemcpy(d_E, h_E, sizeE*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeV*sizeof(long), hipMemcpyHostToDevice);


	//potrebno za BFS
	iteration = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipDeviceSynchronize();

	cout << "pocinje BFS paralelni\n";
	clock_t p1;
	p1 = clock();
	hipEventRecord(start);
	do
	{
		done = true;
		int threadsPerBlock = 256<sizeV ? 256 : sizeV;
		int blocksPerGrid = (sizeV + threadsPerBlock - 1) / threadsPerBlock;
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

		kernel_1 << <blocksPerGrid, threadsPerBlock >> >(d_V, sizeV, d_E, sizeE, d_C);
		
		hipDeviceSynchronize();
		iteration++;
	} while (!done);
	hipEventRecord(stop);
	double diff = (double)(clock() - p1) / CLOCKS_PER_SEC;
	//hipDeviceSynchronize();
	hipEventSynchronize(stop);
	float milliseconds = 0;
	double sec;
	hipEventElapsedTime(&milliseconds, start, stop);
	sec = milliseconds / 1000.0;
	//_log1  << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": event: " << sec << " s\n";
	cout << "Copy C to host\n";
	hipMemcpy(h_C, d_C, sizeV*sizeof(long), hipMemcpyDeviceToHost);
	_log1 << " ; "<< endl;
	copy(h_C, h_C + sizeV, ostream_iterator<long>(_log1, " "));
	_log1 <<  endl;

	cout << "Oslobadjanje memorije\n";
	hipFree(d_E);
	hipFree(d_V);
	hipFree(d_C);

	free(h_C);
	cout << "done" << endl;
	_log1.close();
	return sec;
}

extern "C" double paralelniBFS_1_Share(long *h_V, long *h_E, long sizeV, long sizeE)
{
	cout << "paralelniBFS" << endl;

	cout << "Alokacija host\n";
	long *h_C(NULL);
	h_C = (long*)malloc(sizeV*sizeof(long));
	memset(h_C, 127, sizeV*sizeof(long));

	//pocetne postavke za BFS
	long pocetniCvor = 0;
	//h_F[pocetniCvor] = 1;
	h_C[pocetniCvor] = 0;

	//alokacija na device
	cout << "Alokacija device\n";
	long *d_E(NULL), *d_V(NULL), *d_C(NULL);
	hipMalloc((void**)&d_E, sizeE*sizeof(long));
	hipMalloc((void**)&d_V, sizeV*sizeof(long));
	hipMalloc((void**)&d_C, sizeV*sizeof(long));

	//kopiranje na device
	cout << "Kopiranje na device\n";
	hipMemcpy(d_E, h_E, sizeE*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeV*sizeof(long), hipMemcpyHostToDevice);


	//potrebno za BFS
	iteration = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipDeviceSynchronize();

	cout << "pocinje BFS paralelni\n";
	clock_t p1;
	p1 = clock();
	hipEventRecord(start);
	do
	{
		done = true;
		//doneI = 0;
		int threadsPerBlock = 256<sizeV ? 256 : sizeV;
		int blocksPerGrid = (sizeV + threadsPerBlock - 1) / threadsPerBlock;
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
		//kernel_1_Share<< <blocksPerGrid,threadsPerBlock> >>(d_V, sizeV, d_E, sizeE, d_C);
		kernel_1_Share << <blocksPerGrid, threadsPerBlock >> >(d_V, sizeV, d_E, sizeE, d_C);
		
		hipDeviceSynchronize();
		iteration++;
	} while (!doneI);
	hipEventRecord(stop);
	double diff = (double)(clock() - p1) / CLOCKS_PER_SEC;
	//hipDeviceSynchronize();
	hipEventSynchronize(stop);
	float milliseconds = 0;
	double sec;
	hipEventElapsedTime(&milliseconds, start, stop);
	sec = milliseconds / 1000.0;
	//_log1  << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": event: " << sec << " s\n";
	cout << "Copy C to host\n";
	hipMemcpy(h_C, d_C, sizeV*sizeof(long), hipMemcpyDeviceToHost);
	_log1 << " ; "<< endl;
	copy(h_C, h_C + sizeV, ostream_iterator<long>(_log1, " "));
	_log1 <<  endl;

	cout << "Oslobadjanje memorije\n";
	hipFree(d_E);
	hipFree(d_V);
	hipFree(d_C);

	free(h_C);
	cout << "done" << endl;
	_log1.close();
	return sec;
}

extern "C" double paralelniBFS_1_ShareAtomics(long *h_V, long *h_E, long sizeV, long sizeE)
{
	cout << "paralelniBFS" << endl;

	cout << "Alokacija host\n";
	long *h_C(NULL);
	h_C = (long*)malloc(sizeV*sizeof(long));
	memset(h_C, 127, sizeV*sizeof(long));

	//pocetne postavke za BFS
	long pocetniCvor = 0;
	//h_F[pocetniCvor] = 1;
	h_C[pocetniCvor] = 0;

	//alokacija na device
	cout << "Alokacija device\n";
	long *d_E(NULL), *d_V(NULL), *d_C(NULL);
	hipMalloc((void**)&d_E, sizeE*sizeof(long));
	hipMalloc((void**)&d_V, sizeV*sizeof(long));
	hipMalloc((void**)&d_C, sizeV*sizeof(long));

	//kopiranje na device
	cout << "Kopiranje na device\n";
	hipMemcpy(d_E, h_E, sizeE*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeV*sizeof(long), hipMemcpyHostToDevice);


	//potrebno za BFS
	iteration = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipDeviceSynchronize();

	cout << "pocinje BFS paralelni\n";
	clock_t p1;
	p1 = clock();
	hipEventRecord(start);
	do
	{
		//done = true;
		doneI = 1;
		int threadsPerBlock = 256<sizeV ? 256 : sizeV;
		int blocksPerGrid = (sizeV + threadsPerBlock - 1) / threadsPerBlock;
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
		//kernel_1_Share<< <blocksPerGrid,threadsPerBlock> >>(d_V, sizeV, d_E, sizeE, d_C);
		kernel_1_ShareAtomics<< <blocksPerGrid, threadsPerBlock >> >(d_V, sizeV, d_E, sizeE, d_C);
		
		hipDeviceSynchronize();
		iteration++;
	} while (!doneI);
	hipEventRecord(stop);
	double diff = (double)(clock() - p1) / CLOCKS_PER_SEC;
	//hipDeviceSynchronize();
	hipEventSynchronize(stop);
	float milliseconds = 0;
	double sec;
	hipEventElapsedTime(&milliseconds, start, stop);
	sec = milliseconds / 1000.0;
	//_log1  << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": event: " << sec << " s\n";
	cout << "Copy C to host\n";
	hipMemcpy(h_C, d_C, sizeV*sizeof(long), hipMemcpyDeviceToHost);
	_log1 << " ; " << endl;
	copy(h_C, h_C + sizeV, ostream_iterator<long>(_log1, " "));
	_log1 << endl;

	cout << "Oslobadjanje memorije\n";
	hipFree(d_E);
	hipFree(d_V);
	hipFree(d_C);

	free(h_C);
	cout << "done" << endl;
	_log1.close();
	return sec;
}

extern "C" double paralelniBFSEdge(long *h_V, long *h_E, long sizeV, long sizeE)
{
	cout << "paralelniBFS" << endl;

	cout << "Alokacija host\n";
	long *h_F(NULL), *h_X(NULL), *h_C(NULL);
	h_F = (long*)malloc(sizeV*sizeof(long));
	h_X = (long*)malloc(sizeV*sizeof(long));
	h_C = (long*)malloc(sizeV*sizeof(long));
	memset(h_F, 0, sizeV*sizeof(long));
	memset(h_X, 0, sizeV*sizeof(long));
	memset(h_C, 127, sizeV*sizeof(long));

	//pocetne postavke za BFS
	long pocetniCvor = 0;
	h_F[pocetniCvor] = 1;
	h_C[pocetniCvor] = 0;

	//alokacija na device
	cout << "Alokacija device\n";
	long *d_E(NULL), *d_V(NULL),
		*d_F(NULL), *d_X(NULL), *d_C(NULL);
	hipMalloc((void**)&d_E, sizeE*sizeof(long));
	hipMalloc((void**)&d_V, sizeV*sizeof(long));
	hipMalloc((void**)&d_F, sizeV*sizeof(long));
	hipMalloc((void**)&d_X, sizeV*sizeof(long));
	hipMalloc((void**)&d_C, sizeV*sizeof(long));

	//kopiranje na device
	cout << "Kopiranje na device\n";
	hipMemcpy(d_E, h_E, sizeE*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_F, h_F, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_X, h_X, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeV*sizeof(long), hipMemcpyHostToDevice);


	//potrebno za BFS
	thrust::device_ptr<long> dev_ptr(d_F);
	//bool paralelno = true, seq = false;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipDeviceSynchronize();

	cout << "pocinje BFS paralelni\n";
	clock_t p1;
	p1 = clock();
	hipEventRecord(start);
	while (thrust::reduce(dev_ptr, dev_ptr + sizeV))
	{
		//cout << "While petlja\n";
		int threadsPerBlock = 256<sizeE ? 256 : sizeE;
		int blocksPerGrid = (sizeE + threadsPerBlock - 1) / threadsPerBlock;
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
		edge << <blocksPerGrid, threadsPerBlock >> >(d_V, sizeV, d_E, sizeE, d_F, d_X, d_C);

		//vectorAdd << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, numElements);
		//break;
		//hipDeviceSynchronize();
	}
	hipEventRecord(stop);
	double diff = (double)(clock() - p1) / CLOCKS_PER_SEC;
	//hipDeviceSynchronize();
	hipEventSynchronize(stop);
	float milliseconds = 0;
	double sec;
	hipEventElapsedTime(&milliseconds, start, stop);
	sec = milliseconds / 1000.0;
	//_log1  << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": event: " << sec << " s\n";
	cout << "Copy C to host\n";
	hipMemcpy(h_C, d_C, sizeV*sizeof(long), hipMemcpyDeviceToHost);
	_log1 << " ; " << endl;
	copy(h_C, h_C + sizeV, ostream_iterator<long>(_log1, " "));
	_log1 << endl;

	cout << "Oslobadjanje memorije\n";
	hipFree(d_E);
	hipFree(d_V);
	hipFree(d_F);
	hipFree(d_X);
	hipFree(d_C);

	free(h_F);
	free(h_X);
	free(h_C);

	return sec;
}

extern "C" double paralelniBFS_1_Edge(long *h_V, long *h_E, long sizeV, long sizeE)
{
	cout << "paralelniBFS" << endl;

	cout << "Alokacija host\n";
	long *h_C(NULL);
	h_C = (long*)malloc(sizeV*sizeof(long));
	memset(h_C, 127, sizeV*sizeof(long));

	//pocetne postavke za BFS
	long pocetniCvor = 0;
	//h_F[pocetniCvor] = 1;
	h_C[pocetniCvor] = 0;

	//alokacija na device
	cout << "Alokacija device\n";
	long *d_E(NULL), *d_V(NULL),*d_C(NULL);
	hipMalloc((void**)&d_E, sizeE*sizeof(long));
	hipMalloc((void**)&d_V, sizeV*sizeof(long));
	hipMalloc((void**)&d_C, sizeV*sizeof(long));

	//kopiranje na device
	cout << "Kopiranje na device\n";
	hipMemcpy(d_E, h_E, sizeE*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeV*sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, sizeV*sizeof(long), hipMemcpyHostToDevice);


	//potrebno za BFS
	iteration = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipDeviceSynchronize();

	cout << "pocinje BFS paralelni\n";
	clock_t p1;
	p1 = clock();
	hipEventRecord(start);
	do
	{
		done = true;
		int threadsPerBlock = 256<sizeE ? 256 : sizeE;
		int blocksPerGrid = (sizeE + threadsPerBlock - 1) / threadsPerBlock;
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
		kernel_1_Edge<< <blocksPerGrid, threadsPerBlock >> >(d_V, sizeV, d_E, sizeE, d_C);

		hipDeviceSynchronize();
		iteration++;
	} while (!done);
	hipEventRecord(stop);
	double diff = (double)(clock() - p1) / CLOCKS_PER_SEC;
	//hipDeviceSynchronize();
	hipEventSynchronize(stop);
	float milliseconds = 0;
	double sec;
	hipEventElapsedTime(&milliseconds, start, stop);
	sec = milliseconds / 1000.0;
	//_log1  << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": The time taken for paralel Breadth first search: " << diff << endl;
	cout << endl << ": event: " << sec << " s\n";
	cout << "Copy C to host\n";
	hipMemcpy(h_C, d_C, sizeV*sizeof(long), hipMemcpyDeviceToHost);
	_log1 << " ; " << endl;
	copy(h_C, h_C + sizeV, ostream_iterator<long>(_log1, " "));
	_log1 << endl;

	cout << "Oslobadjanje memorije\n";
	hipFree(d_E);
	hipFree(d_V);
	hipFree(d_C);

	free(h_C);
	cout << "done" << endl;
	_log1.close();
	return sec;
}
